#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
//#include <hip/hip_runtime_api.h>
#include "dot_product.h"

int main(int argc ,char* argv[]) {
	
	FILE *data;
	FILE *vector;
	size_t size;
	size_t sizeV;
	
	/* Initialize rows, cols, CUDA devices and threads from the user */
	unsigned int rows=atoi(argv[3]);
	unsigned int cols=atoi(argv[4]);					  
	int CUDA_DEVICE = atoi(argv[5]);
	int THREADS = atoi(argv[6]);
	
	printf("Rows= %d\n,Cols = %d\n,CUDA_DEVICE= %d\n, THREADS =%d \n",rows,cols,CUDA_DEVICE,THREADS);
	hipError_t err = hipSetDevice(CUDA_DEVICE);

	if(err != hipSuccess) { printf("Error setting CUDA DEVICE\n"); exit(EXIT_FAILURE); }

	/*Host variable declaration */

	//int THREADS = 32;				
	int BLOCKS;
	float* host_results = (float*) malloc(rows * sizeof(float)); 
	struct timeval starttime, endtime;
	clock_t start, end;
	float seconds = 0;
	unsigned int jobs; 
	unsigned long i;

	/*Kernel variable declaration */
	
	float  *dev_dataT;
	float *dev_dataV;
	float *results;
        //size_t len = 0;
	float arr[rows][cols];
	float var ;
	int vrow =1;

	start = clock();

	/* Validation to check if the data file is readable */
	
	data = fopen(argv[1], "r");
	vector = fopen(argv[2],"r");
	
	if (data == NULL)
	{
    		printf("Cannot Open the data ");
		return 0;
	}
	if (vector == NULL)
	{
    		printf("Cannot Open the vector");
		return 0;
	}
	
	size = (size_t)((size_t)rows * (size_t)cols);
	sizeV = (size_t)((size_t)vrow*(size_t)cols);

	printf("Size of the data = %lu\n",size);
	printf("Size of the vector = %lu\n",sizeV);

	fflush(stdout);

	float *dataT = (float*)malloc((size)*sizeof(float));
	float *dataV = (float*)malloc((sizeV)*sizeof(float));

	if(dataT == NULL) {
	        printf("ERROR: Memory for data not allocated.\n");
	}
	if(dataV == NULL) {
	        printf("ERROR: Memory for vector not allocated.\n");
	}
	
        gettimeofday(&starttime, NULL);
	int j = 0;

    /* Transfer the Data from the file to CPU Memory */
	

        for (i =0; i< rows;i++){
		for(j=0; j<cols ; j++){
			fscanf(data,"%f",&var);
                        arr[i][j]=var;
		}
	}
	for (i =0;i<cols;i++){
		for(j= 0; j<rows; j++){
			dataT[rows*i+j]= arr[j][i];
	}
	}		

		for (j=0;j<cols;j++){
			fscanf(vector,"%f",&dataV[j]);
		}
   
	fclose(data);
	fclose(vector);
		printf("Read Data\n");
        fflush(stdout);

        gettimeofday(&endtime, NULL);
        seconds+=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);

        printf("time to read data = %f\n", seconds);

	/* Allocate the Memory in the GPU for data */

        gettimeofday(&starttime, NULL);
	err = hipMalloc((float**) &dev_dataT, (size_t) size * (size_t) sizeof(float));
	if(err != hipSuccess) { printf("Error mallocing data on GPU device\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time for cudamalloc for data =%f\n", seconds);

        gettimeofday(&starttime, NULL);


	/* Allocate the memory in the GPU for vector */
	
        err = hipMalloc((float**) &dev_dataV, sizeV * sizeof(float));
       if(err != hipSuccess) { printf("Error mallocing data on GPU device\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
        printf("time for cudamalloc for vector =%f\n", seconds);

        gettimeofday(&starttime, NULL);
	
	err = hipMalloc((float**) &results, rows * sizeof(float) );
	if(err != hipSuccess) { printf("Error mallocing results on GPU device\n"); }
        gettimeofday(&endtime, NULL); 
seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time for cudamalloc for result =%f\n", seconds);

	/*Copy the data to GPU */
	
	
        gettimeofday(&starttime, NULL);
	err = hipMemcpy(dev_dataT, dataT, (size_t)size *sizeof(float), hipMemcpyHostToDevice);
	if(err != hipSuccess) { printf("Error copying data to GPU\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time to copy  data to GPU=%f\n", seconds);

	
	gettimeofday(&starttime, NULL);
        err = hipMemcpy(dev_dataV, dataV, sizeV*sizeof(float), hipMemcpyHostToDevice);
        if(err != hipSuccess) { printf("Error copying data to GPU\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
        printf("time to copy vector data to GPU=%f\n", seconds);

	jobs = rows;
	BLOCKS = (jobs + THREADS - 1)/THREADS;

        gettimeofday(&starttime, NULL);

	/* Calling the kernel function */
	
	kernel<<<BLOCKS,THREADS>>>(rows,cols,dev_dataT,	dev_dataV, results);
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time for kernel=%f\n", seconds);
		
	/* Copy the results back in host */
	
	hipMemcpy(host_results,results,rows * sizeof(float),hipMemcpyDeviceToHost);
	
	printf("Output of dot product is \n");
	printf("\n");
	
	for(int k = 0; k < jobs; k++) {
		printf("%f ", host_results[k]);
	}
	printf("\n");

	hipFree( dev_dataT );
	hipFree( results );

	end = clock();
	seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("Total time = %f\n", seconds);

	return 0;

}